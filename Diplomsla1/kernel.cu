#include "hip/hip_runtime.h"


/*
1. so vsi proteini enako dolgi ?
2. mutacija
*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <stdlib.h> 
#include <random>

#define maxLenProtein 10
//#define minLenProtein 6
#define numProtein 10
#define Cr 6

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

//nikamor je namjnen zgolj prvemu clenu
enum smer{naprej, nazaj, gor, dol, levo, desno, nikamor};
enum smerobratno{nazaj, naprej, dol, gor, desno, levo, nikamor};
enum vrsta{H, P};

struct tocka {
    int x = 0;
    int y = 0;
    int z = 0;
};

struct protein
{
    int dolzinaProteina = maxLenProtein;
    int proteinSmer[maxLenProtein];
    bool proteinVrsta[maxLenProtein];
// uporabi hash table unorderd set
    struct tocka tocke [maxLenProtein];
    int hevristika = 0;
};

__global__ void addKernel(int *c, struct protein* Arr)
{
    int i = threadIdx.x;
    c[i] = Arr[i].proteinSmer[1];
}

bool enako(struct protein ena, struct protein dva) {
    bool enak = true;
    for (int i = 0; i < maxLenProtein; i++) {
        if (ena.proteinSmer[i] != dva.proteinSmer[i]) {
            return false;
        }
    }
    return true;
}

void tvori_mrezo(struct protein &prot) {
    for (int i = 0; i < maxLenProtein; i++) {
        struct tocka t;
        if (i != 0) {
            t = prot.tocke[i - 1];
        }
        
        if (prot.proteinSmer[i] == nikamor) {
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == naprej) {
            t.x++;
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == nazaj) {
            t.x--;
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == gor) {
            t.z++;
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == dol) {
            t.z--;
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == levo) {
            t.y++;
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == desno) {
            t.y--;
            prot.tocke[i] = t;
        }
    }
}

bool cikelj(struct tocka a, struct tocka b) {
    if (a.x == b.x && a.y == b.y && a.z == b.z) {
        return true;
    }
    return false;
}

bool hevristicna_povezava(struct tocka a, struct tocka b) {

    if (a.x == b.x && a.y == b.y && (a.z == b.z + 1 || a.z == b.z - 1)) {
        return true;
    }
    if (a.x == b.x && a.z == b.z && (a.y == b.y + 1 || a.y == b.y - 1)) {
        return true;
    }
    if (a.z == b.z && a.y == b.y && (a.x == b.x + 1 || a.x == b.x - 1)) {
        return true;
    }
    //preverjat za nazaj povezave
    return false;
}

void doloci_hevristiko(struct protein &prot) {
    int br = 0;
    for (int i = 0; i < maxLenProtein-3; i++) {

        if (br == 1) {
            break;
        }

        for (int j = i+3; j < maxLenProtein; j++) {
            if (cikelj(prot.tocke[i], prot.tocke[j])) {
                prot.hevristika = INT_MAX;
                // v primeru cikla minus dolzina proteina
                br = 1;
            }
            if (br == 1) {
                break;
            }

            if (hevristicna_povezava(prot.tocke[i], prot.tocke[j])) {
                prot.hevristika++;
            }
        }
    }
}

int main()
{
    // dolzina proteina 5 - 256

    //struct protein *arr = new struct protein[numProtein];
    struct protein arr1[numProtein];
    srand(time(NULL));
    for (unsigned int i = 0; i < numProtein; i++) {
        struct protein generiranProtein;
        //generiranProtein.dolzinaProteina = rand() %(maxLenProtein - minLenProtein) + minLenProtein;
        int prev = 20;
        for (unsigned int j = 0; j < generiranProtein.dolzinaProteina; j++) {
            bool running = true;
            while (true) {
                generiranProtein.proteinSmer[j] = rand() % 6;
                if(prev == )
                prev = generiranProtein.proteinSmer[j] = rand() % 6;
            }
            prev = generiranProtein.proteinSmer[j];
            generiranProtein.proteinVrsta[j] = rand() % 2;
        }
        arr1[i] = generiranProtein;
    }

    arr1[0].proteinSmer[0] = 6;
    while (true) {
        for (int i = 0; i < numProtein; i++) {

            // KRIZANJE
            int r1 = rand() % numProtein;

            while (r1 == i)
            {
                r1 = rand() % numProtein;
            }

            struct protein generiranProtein1;
            struct protein generiranProtein2;

            int crosoverPoint = rand() % 10;

            for (int j = 0; j < maxLenProtein; j++)
            {
                if (crosoverPoint < j)
                {
                    // memory copy std::copy
                    generiranProtein1.proteinSmer[j] = arr1[i].proteinSmer[j];
                    generiranProtein1.proteinVrsta[j] = arr1[i].proteinVrsta[j];
                    generiranProtein2.proteinSmer[j] = arr1[r1].proteinSmer[j];
                    generiranProtein2.proteinVrsta[j] = arr1[r1].proteinVrsta[j];

                }
                else {
                    generiranProtein1.proteinSmer[j] = arr1[r1].proteinSmer[j];
                    generiranProtein1.proteinVrsta[j] = arr1[r1].proteinVrsta[j];
                    generiranProtein2.proteinSmer[j] = arr1[i].proteinSmer[j];
                    generiranProtein2.proteinVrsta[j] = arr1[i].proteinVrsta[j];
                }
            }

            if (enako(generiranProtein1, arr1[i]) || enako(generiranProtein1, arr1[r1]) || rand() % 100 < 1) {
                int tockaObrta1 = rand() % maxLenProtein;
                int tockaObrta2 = rand() % maxLenProtein;
                while (tockaObrta1 == tockaObrta2) {
                    tockaObrta2 = rand() % maxLenProtein;
                }
                // generiram 3-5 smeri in jih nastavim naklju�no po proteinu
                int temp = generiranProtein1.proteinSmer[tockaObrta1];
                generiranProtein1.proteinSmer[tockaObrta1] = generiranProtein1.proteinSmer[tockaObrta2];
                generiranProtein1.proteinSmer[tockaObrta2] = temp;
            }
            if (enako(generiranProtein2, arr1[i]) || enako(generiranProtein2, arr1[r1]) || rand() % 100 < 1) {
                int tockaObrta1 = rand() % maxLenProtein;
                int tockaObrta2 = rand() % maxLenProtein;
                while (tockaObrta1 == tockaObrta2) {
                    tockaObrta2 = rand() % maxLenProtein;
                }
                int temp = generiranProtein1.proteinSmer[tockaObrta1];
                generiranProtein1.proteinSmer[tockaObrta1] = generiranProtein1.proteinSmer[tockaObrta2];
                generiranProtein1.proteinSmer[tockaObrta2] = temp;
            }

            tvori_mrezo(generiranProtein1);
            tvori_mrezo(generiranProtein2);
            tvori_mrezo(arr1[i]);

            doloci_hevristiko(generiranProtein1);
            doloci_hevristiko(generiranProtein2);
            doloci_hevristiko(arr1[i]);


        }
 }
   













    int c[10] = {0};

    printf(" {%d,%d,%d,%d,%d,%d,%d}\n",
        arr1[1].proteinSmer[1], arr1[2].proteinSmer[1], arr1[3].proteinSmer[1], arr1[4].proteinSmer[1], arr1[5].proteinSmer[1], arr1[6].proteinSmer[1], arr1[7].proteinSmer[1]);
    printf(" {%d,%d,%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4], c[5], c[6]);
    // Add vectors in parallel.
    int* dev_b = 0;
    struct protein* arrr = 0;
    hipSetDevice(0);
    hipMalloc((void**)&dev_b, 10 * sizeof(int));
    hipMalloc((void**)&arrr, 10 * sizeof(struct protein));
    hipMemcpy(arrr, arr1, 10 * sizeof(struct protein), hipMemcpyHostToDevice);
    addKernel <<<1, 10 >>> (dev_b, arrr);

    hipMemcpy(c, dev_b, 10 * sizeof(int), hipMemcpyDeviceToHost);
    printf(" {%d,%d,%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4], c[5], c[6]);

    hipFree(c);
    hipFree(arrr);
    hipFree(dev_b);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.

    