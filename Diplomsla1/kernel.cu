

/*
1. so vsi proteini enako dolgi ?
2. mutacija
*/
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <time.h>
#include <stdlib.h> 
#include <random>

#define maxLenProtein 10
//#define minLenProtein 6
#define numProtein 10
#define Cr 6

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

//nikamor je namjnen zgolj prvemu clenu
enum smer{naprej, nazaj, gor, dol, levo, desno, nikamor};
enum vrsta{H, P};

struct tocka {
    int x = 0;
    int y = 0;
    int z = 0;
};

struct protein
{
    int dolzinaProteina = maxLenProtein;
    int proteinSmer[maxLenProtein];
    bool proteinVrsta[maxLenProtein];
    struct tocka tocke [maxLenProtein];
    int hevristika;
};

__global__ void addKernel(int *c, struct protein* Arr)
{
    int i = threadIdx.x;
    c[i] = Arr[i].proteinSmer[1];
}

bool enako(struct protein ena, struct protein dva) {
    bool enak = true;
    for (int i = 0; i < maxLenProtein; i++) {
        if (ena.proteinSmer[i] != dva.proteinSmer[i]) {
            return false;
        }
    }
    return true;
}

void tvori_mrezo(struct protein prot) {
    for (int i = 0; i < maxLenProtein; i++) {
        struct tocka t;
        if (i != 0) {
            t = prot.tocke[i - 1];
        }
        
        if (prot.proteinSmer[i] == nikamor) {
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == naprej) {
            t.x++;
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == nazaj) {
            t.x--;
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == gor) {
            t.z++;
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == dol) {
            t.z--;
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == levo) {
            t.y++;
        }
        if (prot.proteinSmer[i] == desno) {
            t.y--;
            prot.tocke[i] = t;
        }
    }
}

void doloci_hevristiko(struct protein prot) {

}

int main()
{
    
    //struct protein *arr = new struct protein[numProtein];
    struct protein arr1[numProtein];
    srand(time(NULL));
    for (unsigned int i = 0; i < numProtein; i++) {
        struct protein generiranProtein;
        //generiranProtein.dolzinaProteina = rand() %(maxLenProtein - minLenProtein) + minLenProtein;

        for (unsigned int j = 0; j < generiranProtein.dolzinaProteina; j++) {
            generiranProtein.proteinSmer[j] = rand() % 6;
            generiranProtein.proteinVrsta[j] = rand() % 2;
        }
        arr1[i] = generiranProtein;
    }

    arr1[0].proteinSmer[0] = 6;
 
    for (int i = 0; i < numProtein; i++) {

        // KRIZANJE
        int r1 = rand() % numProtein;

        while (r1 == i)
        {
            r1 = rand() % numProtein;
        }

        struct protein generiranProtein1;
        struct protein generiranProtein2;

        int crosoverPoint = rand() % 10;

        for (int j = 0; j < maxLenProtein; j++)
        {
            if (crosoverPoint < j )
            {
                generiranProtein1.proteinSmer[j] = arr1[i].proteinSmer[j];
                generiranProtein1.proteinVrsta[j] = arr1[i].proteinVrsta[j];
                generiranProtein2.proteinSmer[j] = arr1[r1].proteinSmer[j];
                generiranProtein2.proteinVrsta[j] = arr1[r1].proteinVrsta[j];

            }
            else {
                generiranProtein1.proteinSmer[j] = arr1[r1].proteinSmer[j];
                generiranProtein1.proteinVrsta[j] = arr1[r1].proteinVrsta[j];
                generiranProtein2.proteinSmer[j] = arr1[i].proteinSmer[j];
                generiranProtein2.proteinVrsta[j] = arr1[i].proteinVrsta[j];
            }
        }
        if (enako(generiranProtein1, arr1[i]) || enako(generiranProtein1, arr1[r1]) || rand() % 100 < 1) {
            int tockaObrta1 = rand() % maxLenProtein;
            int tockaObrta2 = rand() % maxLenProtein;
            while (tockaObrta1 == tockaObrta2) {
                tockaObrta2 = rand() % maxLenProtein;
            }
            int temp = generiranProtein1.proteinSmer[tockaObrta1];
            generiranProtein1.proteinSmer[tockaObrta1] = generiranProtein1.proteinSmer[tockaObrta2];
            generiranProtein1.proteinSmer[tockaObrta2] = temp;
        }
        if (enako(generiranProtein2, arr1[i]) || enako(generiranProtein2, arr1[r1]) || rand() % 100 < 1) {
            int tockaObrta1 = rand() % maxLenProtein;
            int tockaObrta2 = rand() % maxLenProtein;
            while (tockaObrta1 == tockaObrta2) {
                tockaObrta2 = rand() % maxLenProtein;
            }
            int temp = generiranProtein1.proteinSmer[tockaObrta1];
            generiranProtein1.proteinSmer[tockaObrta1] = generiranProtein1.proteinSmer[tockaObrta2];
            generiranProtein1.proteinSmer[tockaObrta2] = temp;
        }

        tvori_mrezo(generiranProtein1);
        tvori_mrezo(generiranProtein2);
        tvori_mrezo(arr1[i]);

        doloci_hevristiko(generiranProtein1);
        doloci_hevristiko(generiranProtein2);
        doloci_hevristiko(arr1[i]);


    }













    int c[10] = {0};

    printf(" {%d,%d,%d,%d,%d,%d,%d}\n",
        arr1[1].proteinSmer[1], arr1[2].proteinSmer[1], arr1[3].proteinSmer[1], arr1[4].proteinSmer[1], arr1[5].proteinSmer[1], arr1[6].proteinSmer[1], arr1[7].proteinSmer[1]);
    printf(" {%d,%d,%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4], c[5], c[6]);
    // Add vectors in parallel.
    int* dev_b = 0;
    struct protein* arrr = 0;
    hipSetDevice(0);
    hipMalloc((void**)&dev_b, 10 * sizeof(int));
    hipMalloc((void**)&arrr, 10 * sizeof(struct protein));
    hipMemcpy(arrr, arr1, 10 * sizeof(struct protein), hipMemcpyHostToDevice);
    addKernel <<<1, 10 >>> (dev_b, arrr);

    hipMemcpy(c, dev_b, 10 * sizeof(int), hipMemcpyDeviceToHost);
    printf(" {%d,%d,%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4], c[5], c[6]);

    hipFree(c);
    hipFree(arrr);
    hipFree(dev_b);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.

    