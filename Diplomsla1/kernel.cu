

/*
1. so vsi proteini enako dolgi ?
2. mutacija
*/
#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <time.h>
#include <stdlib.h> 
#include <random>
#include <algorithm>
// dolzina proteina 5 - 256
#define maxLenProtein 256
#define minLenProtein 5
#define numProtein 210
#define Cr 6

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

//nikamor je namjnen zgolj prvemu clenu
enum smer{nikamor, naprej, nazaj, gor, dol, levo, desno};
//enum smerobratno{nikamor, nazaj, naprej, dol, gor, desno, levo};
enum vrsta{H, P};

struct tocka {
    int x = 0;
    int y = 0;
    int z = 0;
};

struct protein
{
    int dolzinaProteina = maxLenProtein;
    int proteinSmer[maxLenProtein] = {0};
    bool proteinVrsta[maxLenProtein];
    struct tocka tocke [maxLenProtein];
    int hevristika = 0;
};

__global__ void addKernel(int *c, struct protein* Arr)
{
    int i = threadIdx.x;
    c[i] = Arr[i].proteinSmer[1];
}

bool enako(struct protein ena, struct protein dva) {
    bool enak = true;
    for (int i = 0; i < maxLenProtein; i++) {
        if (ena.proteinSmer[i] != dva.proteinSmer[i]) {
            return false;
        }
    }
    return true;
}

void tvori_mrezo(struct protein &prot) {
    for (int i = 0; i < maxLenProtein; i++) {
        struct tocka t;
        if (i != 0) {
            t = prot.tocke[i - 1];
        }
        
        if (prot.proteinSmer[i] == nikamor) {
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == naprej) {
            t.x++;
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == nazaj) {
            t.x--;
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == gor) {
            t.z++;
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == dol) {
            t.z--;
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == levo) {
            t.y++;
            prot.tocke[i] = t;
        }
        if (prot.proteinSmer[i] == desno) {
            t.y--;
            prot.tocke[i] = t;
        }
    }
}

bool cikelj(struct tocka a, struct tocka b) {
    if (a.x == b.x && a.y == b.y && a.z == b.z) {
        return true;
    }
    return false;
}

bool hevristicna_povezava(struct tocka a, struct tocka b) {

    //povezav za nazaj ni potrebno preverjati, saj tocke v katerih se isce pocvezava so vsaj 3 narazen
    if (a.x == b.x && a.y == b.y && (a.z == b.z + 1 || a.z == b.z - 1)) {
        return true;
    }
    if (a.x == b.x && a.z == b.z && (a.y == b.y + 1 || a.y == b.y - 1)) {
        return true;
    }
    if (a.z == b.z && a.y == b.y && (a.x == b.x + 1 || a.x == b.x - 1)) {
        return true;
    }

    return false;
}

void doloci_hevristiko(struct protein &prot) {
    int br = 0;
    prot.hevristika = 0;
    for (int i = 0; i < maxLenProtein-3; i++) {

        for (int j = i+3; j < maxLenProtein; j++) {
            if (cikelj(prot.tocke[i], prot.tocke[j])) {
                // v primeru cikla minus dolzina proteina
                prot.hevristika -= prot.dolzinaProteina;
            }

            if (prot.proteinVrsta[i] == H && prot.proteinVrsta[j] == H) {
                if (hevristicna_povezava(prot.tocke[i], prot.tocke[j])) {
                    prot.hevristika++;
                }
            }
        }
    }
}

int main()
{
    //struct protein *arr = new struct protein[numProtein];
    int nfes = 0;
    std::cin >> nfes;
    nfes *= 1000;
    int count = 0;
    struct protein arr1[numProtein];
    srand(5);
    for (int i = 0; i < numProtein; i++) {
        struct protein generiranProtein ;
        // generiramo dolzino proteina
        generiranProtein.dolzinaProteina = rand() %(maxLenProtein - minLenProtein) + minLenProtein;
        int prev = 20;
        int premik = 0;
        //tvorimo smeri proteina pri cemer preverjamo da netvorimo kontra strani od prejsne
        for ( int j = 0; j < generiranProtein.dolzinaProteina; j++) {

            generiranProtein.proteinSmer[j] = rand() % 6+1;
            if (prev % 2 == 0) {
                premik = -1;
            }
            else {
                premik = 1;
            }
            while (prev + premik == generiranProtein.proteinSmer[j]) {
               generiranProtein.proteinSmer[j] = rand() % 6+1;
            }
            prev = generiranProtein.proteinSmer[j];


            generiranProtein.proteinVrsta[j] = rand() % 2;
        }
        arr1[i] = generiranProtein;
    }

    while (count < nfes) {

        for (int i = 0; i < numProtein; i++) {

            // KRIZANJE
            int r1 = rand() % numProtein;

            while (r1 == i)
            {
                r1 = rand() % numProtein;
            }

            struct protein generiranProtein1;
            struct protein generiranProtein2; 
            
            struct protein najdenI = arr1[i];
            struct protein najdenR1 = arr1[r1];

            int crospoint = 0;
            if (najdenI.dolzinaProteina < najdenR1.dolzinaProteina) {
                crospoint = najdenI.dolzinaProteina;
            }
            else {
                crospoint = najdenR1.dolzinaProteina;
            }
            crospoint -= 3;
            int crosoverPoint = rand() % crospoint + 3;

            
            std::copy(najdenI.proteinSmer, najdenI.proteinSmer + crosoverPoint, generiranProtein1.proteinSmer);
            std::copy(najdenR1.proteinSmer + crosoverPoint, najdenR1.proteinSmer + najdenR1.dolzinaProteina, generiranProtein1.proteinSmer + crosoverPoint);

            std::copy(najdenI.proteinVrsta, najdenI.proteinVrsta + crosoverPoint, generiranProtein1.proteinVrsta);
            std::copy(najdenR1.proteinVrsta + crosoverPoint, najdenR1.proteinVrsta + najdenR1.dolzinaProteina, generiranProtein1.proteinVrsta + crosoverPoint);

            generiranProtein1.dolzinaProteina = najdenR1.dolzinaProteina;
            

            std::copy(najdenR1.proteinSmer, najdenR1.proteinSmer + crosoverPoint, generiranProtein2.proteinSmer);
            std::copy(najdenI.proteinSmer + crosoverPoint, najdenI.proteinSmer + najdenI.dolzinaProteina, generiranProtein2.proteinSmer + crosoverPoint);

            std::copy(najdenR1.proteinVrsta, najdenR1.proteinVrsta + crosoverPoint, generiranProtein2.proteinVrsta);
            std::copy(najdenI.proteinVrsta + crosoverPoint, najdenI.proteinVrsta + najdenI.dolzinaProteina, generiranProtein2.proteinVrsta + crosoverPoint);

            generiranProtein2.dolzinaProteina = najdenI.dolzinaProteina;
            /*
            for (int j = 0; j < maxLenProtein; j++)
            {
                if (crosoverPoint < j)
                {
                    // memory copy std::copy
                    generiranProtein1.proteinSmer[j] = arr1[i].proteinSmer[j];
                    generiranProtein1.proteinVrsta[j] = arr1[i].proteinVrsta[j];
                     generiranProtein2.dolzinaProteina = arr1[i].dolzinaProteina;generiranProtein2.dolzinaProteina = arr1[i].dolzinaProteina;
                    generiranProtein2.proteinSmer[j] = arr1[r1].proteinSmer[j];
                    generiranProtein2.proteinVrsta[j] = arr1[r1].proteinVrsta[j];

                }
                else {
                    generiranProtein1.proteinSmer[j] = arr1[r1].proteinSmer[j];
                    generiranProtein1.proteinVrsta[j] = arr1[r1].proteinVrsta[j];
                    generiranProtein2.dolzinaProteina = arr1[i].dolzinaProteina;generiranProtein2.dolzinaProteina = arr1[i].dolzinaProteina;
                    generiranProtein2.proteinSmer[j] = arr1[i].proteinSmer[j];
                    generiranProtein2.proteinVrsta[j] = arr1[i].proteinVrsta[j];
                }
            }
            */
            if (enako(generiranProtein1, najdenI) || enako(generiranProtein1, najdenR1) || rand() % 100 < 2) {

                int tockaObrta1 = rand() % generiranProtein1.dolzinaProteina;
                int tockaObrta2 = rand() % generiranProtein1.dolzinaProteina;
                int tockaObrta3 = rand() % generiranProtein1.dolzinaProteina;
                int premik = 0;

                generiranProtein1.proteinSmer[tockaObrta1] = rand() % 6+1;
                if (generiranProtein1.proteinSmer[tockaObrta1 - 1] % 2 == 0) {
                    premik = -1;
                }
                else {
                    premik = 1;
                }
                while (generiranProtein1.proteinSmer[tockaObrta1 - 1] + premik == generiranProtein1.proteinSmer[tockaObrta1]) {
                    generiranProtein1.proteinSmer[tockaObrta1 - 1] = rand() % 6+1;
                }

                generiranProtein1.proteinSmer[tockaObrta2] = rand() % 6+1;
                if (generiranProtein1.proteinSmer[tockaObrta2 - 1] % 2 == 0) {
                    premik = -1;
                }
                else {
                    premik = 1;
                }
                while (generiranProtein1.proteinSmer[tockaObrta2 - 1] + premik == generiranProtein1.proteinSmer[tockaObrta2]) {
                    generiranProtein1.proteinSmer[tockaObrta2 - 1] = rand() % 6+1;
                }

                generiranProtein1.proteinSmer[tockaObrta3] = rand() % 6+1;
                if (generiranProtein1.proteinSmer[tockaObrta3 - 1] % 2 == 0) {
                    premik = -1;
                }
                else {
                    premik = 1;
                }
                while (generiranProtein1.proteinSmer[tockaObrta3 - 1] + premik == generiranProtein1.proteinSmer[tockaObrta3]) {
                    generiranProtein1.proteinSmer[tockaObrta3 - 1] = rand() % 6+1;
                }

                generiranProtein1.proteinVrsta[tockaObrta1] = rand() % 2;
                generiranProtein1.proteinVrsta[tockaObrta2] = rand() % 2;
                generiranProtein1.proteinVrsta[tockaObrta3] = rand() % 2;

            }
            if (enako(generiranProtein2, najdenI) || enako(generiranProtein2, najdenR1) || rand() % 100 < 1) {
                int tockaObrta1 = rand() % generiranProtein2.dolzinaProteina;
                int tockaObrta2 = rand() % generiranProtein2.dolzinaProteina;
                int tockaObrta3 = rand() % generiranProtein2.dolzinaProteina;
                int premik = 0;
                generiranProtein2.proteinSmer[tockaObrta1] = rand() % 6+1;
                if (generiranProtein2.proteinSmer[tockaObrta1 - 1] % 2 == 0) {
                    premik = -1;
                }
                else {
                    premik = 1;
                }
                while (generiranProtein2.proteinSmer[tockaObrta1 - 1] + premik == generiranProtein2.proteinSmer[tockaObrta1]) {
                    generiranProtein2.proteinSmer[tockaObrta1 - 1] = rand() % 6+1;
                }

                generiranProtein2.proteinSmer[tockaObrta2] = rand() % 6+1;
                if (generiranProtein2.proteinSmer[tockaObrta2 - 1] % 2 == 0) {
                    premik = -1;
                }
                else {
                    premik = 1;
                }
                while (generiranProtein2.proteinSmer[tockaObrta2 - 1] + premik == generiranProtein2.proteinSmer[tockaObrta2]) {
                    generiranProtein2.proteinSmer[tockaObrta2 - 1] = rand() % 6+1;
                }

                generiranProtein2.proteinSmer[tockaObrta3] = rand() % 6+1;
                if (generiranProtein2.proteinSmer[tockaObrta3 - 1] % 2 == 0) {
                    premik = -1;
                }
                else {
                    premik = 1;
                }
                while (generiranProtein2.proteinSmer[tockaObrta3 - 1] + premik == generiranProtein2.proteinSmer[tockaObrta3]) {
                    generiranProtein2.proteinSmer[tockaObrta3 - 1] = rand() % 6+1;
                }

                generiranProtein2.proteinVrsta[tockaObrta1] = rand() % 2;
                generiranProtein2.proteinVrsta[tockaObrta2] = rand() % 2;
                generiranProtein2.proteinVrsta[tockaObrta3] = rand() % 2;

            }

            tvori_mrezo(generiranProtein1);
            tvori_mrezo(generiranProtein2);
            tvori_mrezo(najdenI);

            doloci_hevristiko(generiranProtein1);
            doloci_hevristiko(generiranProtein2);
            doloci_hevristiko(najdenI);
            
            if ( generiranProtein1.hevristika > generiranProtein2.hevristika) {
                if (najdenI.hevristika < generiranProtein1.hevristika) {
                    arr1[i] = generiranProtein1;
                }
            }
            else {
                if (najdenI.hevristika < generiranProtein2.hevristika) {
                    arr1[i] = generiranProtein2;
                }
            }

            count++;
        }
    }
   













    int c[10] = {0};

    printf(" {%d,%d,%d,%d,%d,%d,%d}\n",
        arr1[1].proteinSmer[1], arr1[2].proteinSmer[1], arr1[3].proteinSmer[1], arr1[4].proteinSmer[1], arr1[5].proteinSmer[1], arr1[6].proteinSmer[1], arr1[7].proteinSmer[1]);
    printf(" {%d,%d,%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4], c[5], c[6]);
    // Add vectors in parallel.
    int* dev_b = 0;
    struct protein* arrr = 0;
    hipSetDevice(0);
    hipMalloc((void**)&dev_b, 10 * sizeof(int));
    hipMalloc((void**)&arrr, 10 * sizeof(struct protein));
    hipMemcpy(arrr, arr1, 10 * sizeof(struct protein), hipMemcpyHostToDevice);
    addKernel <<<1, 10 >>> (dev_b, arrr);

    hipMemcpy(c, dev_b, 10 * sizeof(int), hipMemcpyDeviceToHost);
    printf(" {%d,%d,%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4], c[5], c[6]);

    hipFree(c);
    hipFree(arrr);
    hipFree(dev_b);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.

    